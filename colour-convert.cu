#include "hip/hip_runtime.h"
#include <stdio.h>
#include <string.h>
#include <stdlib.h>
#include <helper_timer.h>
#include "colour-convert.h"

// number of threads
// change it for experiment
int T = 1000;

// load empty kernal
__global__ void mykernel(void)
{
    
}



__global__ void rgb2yuvKernel(unsigned char *imgr,unsigned char *imgg,unsigned char *imgb,unsigned char *imgy,unsigned char *imgcb,unsigned char *imgcr) {

    unsigned char r, g, b;
    unsigned char y, cb, cr;

    int index;
    index = threadIdx.x + blockIdx.x * blockDim.x;
   

    r = imgr[index];
    g = imgg[index];
    b = imgb[index];
    
    y  = (unsigned char)( 0.299*r + 0.587*g +  0.114*b);
    cb = (unsigned char)(-0.169*r - 0.331*g +  0.499*b + 128);
    cr = (unsigned char)( 0.499*r - 0.418*g - 0.0813*b + 128);
    
    imgy[index] = y;
    imgcb[index] = cb;
    imgcr[index] = cr;
    

}
 
 
__global__ void yuv2rgbKernel(unsigned char *imgr,unsigned char *imgg,unsigned char *imgb,unsigned char *imgy,unsigned char *imgcb,unsigned char *imgcr) {

    int  rt,gt,bt;
    int y, cb, cr;    
    int index;
    index = threadIdx.x + blockIdx.x * blockDim.x;


    y  = (int)imgy[index];
    cb = (int)imgcb[index] - 128;
    cr = (int)imgcr[index] - 128;
    
    rt  = (int)( y + 1.402*cr);
    if(rt > 255)
        rt = 255;
    if(rt < 0)
        rt =  0;
    gt  = (int)( y - 0.344*cb - 0.714*cr); 
    if(gt > 255)
        gt =  255;
    if(gt < 0)
        gt =  0;           
    bt  = (int)( y + 1.772*cb);
    if(bt > 255)
        bt =  255;
    if(bt < 0)
        bt = 0;


    imgr[index] = rt;
    imgg[index] = gt;
    imgb[index] = bt;
}




PPM_IMG copy_and_return_PPM(PPM_IMG img_in)
{
    PPM_IMG img_out;
    StopWatchInterface *timer=NULL;

    int TOTAL_PIXEL = img_in.w*img_in.h;

    int size = TOTAL_PIXEL * sizeof(char);


    img_out.w = img_in.w;
    img_out.h = img_in.h;
    img_out.img_r = (unsigned char *)malloc(size);
    img_out.img_g = (unsigned char *)malloc(size);
    img_out.img_b = (unsigned char *)malloc(size);

    //Put you CUDA initialization code here.
    unsigned char *r_d, *g_d, *b_d;
    unsigned char *rr_d, *gg_d, *bb_d;
    
    hipMalloc((void **)&r_d, size);
    hipMalloc((void **)&g_d, size);
    hipMalloc((void **)&b_d, size);

    hipMalloc((void **)&rr_d, size);
    hipMalloc((void **)&gg_d, size);
    hipMalloc((void **)&bb_d, size);

    sdkCreateTimer(&timer);
    sdkStartTimer(&timer);

    // Copy RGB inputs from host to device
    hipMemcpy(r_d, img_in.img_r, size, hipMemcpyHostToDevice);
    hipMemcpy(g_d, img_in.img_g, size, hipMemcpyHostToDevice);
    hipMemcpy(b_d, img_in.img_b, size, hipMemcpyHostToDevice);

    sdkStopTimer(&timer);

    printf("Copy time(Host to Device): %f (ms)\n", sdkGetTimerValue(&timer));
    sdkDeleteTimer(&timer);

   
    sdkCreateTimer(&timer);
    sdkStartTimer(&timer);

    // Copy YUV output from device to host
    hipMemcpy(img_out.img_r, r_d, size, hipMemcpyDeviceToHost);
    hipMemcpy(img_out.img_g, g_d, size, hipMemcpyDeviceToHost);
    hipMemcpy(img_out.img_b, b_d, size, hipMemcpyDeviceToHost);

    sdkStopTimer(&timer);
    printf("Copy time(Device to Host): %f (ms)\n", sdkGetTimerValue(&timer));
    sdkDeleteTimer(&timer);

    // freeing intermediate memory
    hipFree(r_d);hipFree(g_d);hipFree(b_d);hipFree(rr_d);hipFree(gg_d);hipFree(bb_d);

    return img_out;    
}




// we call this function to load empty kernel 
void load_empty_kernel()
{
     mykernel<<<1,1>>>();
}



YUV_IMG rgb2yuvGPU(PPM_IMG img_in)
{

    YUV_IMG img_out;
    //Put you CUDA initialization code here.
    
    unsigned char *d_r, *d_g, *d_b;
    unsigned char *d_y, *d_cb, *d_cr;

    img_out.w = img_in.w;
    img_out.h = img_in.h;


    img_out.img_y = (unsigned char *)malloc(sizeof(unsigned char)*img_out.w*img_out.h);
    img_out.img_u = (unsigned char *)malloc(sizeof(unsigned char)*img_out.w*img_out.h);
    img_out.img_v = (unsigned char *)malloc(sizeof(unsigned char)*img_out.w*img_out.h);


    hipMalloc((void **)&d_r, sizeof(unsigned char)*img_out.w*img_out.h);
    hipMalloc((void **)&d_g, sizeof(unsigned char)*img_out.w*img_out.h);
    hipMalloc((void **)&d_b, sizeof(unsigned char)*img_out.w*img_out.h);

    
    hipMalloc((void **)&d_y, sizeof(unsigned char)*img_out.w*img_out.h);
    hipMalloc((void **)&d_cb, sizeof(unsigned char)*img_out.w*img_out.h);
    hipMalloc((void **)&d_cr, sizeof(unsigned char)*img_out.w*img_out.h);   


    hipMemcpy(d_r, img_in.img_r, sizeof(unsigned char)*img_out.w*img_out.h, hipMemcpyHostToDevice);
    hipMemcpy(d_g, img_in.img_g, sizeof(unsigned char)*img_out.w*img_out.h, hipMemcpyHostToDevice);
    hipMemcpy(d_b, img_in.img_b, sizeof(unsigned char)*img_out.w*img_out.h, hipMemcpyHostToDevice);
    

    rgb2yuvKernel<<<(img_in.w*img_in.h)/T,T>>>(d_r,d_g,d_b,d_y,d_cb,d_cr);//Launch the Kernel


    hipMemcpy(img_out.img_y, d_y, sizeof(unsigned char)*img_out.w*img_out.h, hipMemcpyDeviceToHost);
    hipMemcpy(img_out.img_u, d_cb, sizeof(unsigned char)*img_out.w*img_out.h, hipMemcpyDeviceToHost);
    hipMemcpy(img_out.img_v, d_cr, sizeof(unsigned char)*img_out.w*img_out.h, hipMemcpyDeviceToHost);

    hipFree(d_r);hipFree(d_g);hipFree(d_b);hipFree(d_y);hipFree(d_cb);hipFree(d_cr);
    return img_out;
}




PPM_IMG yuv2rgbGPU(YUV_IMG img_in)
{
    PPM_IMG img_out;
    //Put you CUDA setup code here.

    unsigned char *d_r, *d_g, *d_b;
    unsigned char *d_y, *d_cb, *d_cr;

    img_out.w = img_in.w;
    img_out.h = img_in.h;


    img_out.img_r = (unsigned char *)malloc(sizeof(unsigned char)*img_out.w*img_out.h);
    img_out.img_g = (unsigned char *)malloc(sizeof(unsigned char)*img_out.w*img_out.h);
    img_out.img_b = (unsigned char *)malloc(sizeof(unsigned char)*img_out.w*img_out.h);


    hipMalloc((void **)&d_r, sizeof(unsigned char)*img_out.w*img_out.h);
    hipMalloc((void **)&d_g, sizeof(unsigned char)*img_out.w*img_out.h);
    hipMalloc((void **)&d_b, sizeof(unsigned char)*img_out.w*img_out.h);

    
    hipMalloc((void **)&d_y, sizeof(unsigned char)*img_out.w*img_out.h);
    hipMalloc((void **)&d_cb, sizeof(unsigned char)*img_out.w*img_out.h);
    hipMalloc((void **)&d_cr, sizeof(unsigned char)*img_out.w*img_out.h);   


    hipMemcpy(d_y, img_in.img_y, sizeof(unsigned char)*img_out.w*img_out.h, hipMemcpyHostToDevice);
    hipMemcpy(d_cb, img_in.img_u, sizeof(unsigned char)*img_out.w*img_out.h, hipMemcpyHostToDevice);
    hipMemcpy(d_cr, img_in.img_v, sizeof(unsigned char)*img_out.w*img_out.h, hipMemcpyHostToDevice);
    

    yuv2rgbKernel<<<(img_in.w*img_in.h)/T,T>>>(d_r,d_g,d_b,d_y,d_cb,d_cr);//Launch the Kernel


    hipMemcpy(img_out.img_r, d_r, sizeof(unsigned char)*img_out.w*img_out.h, hipMemcpyDeviceToHost);
    hipMemcpy(img_out.img_g, d_g, sizeof(unsigned char)*img_out.w*img_out.h, hipMemcpyDeviceToHost);
    hipMemcpy(img_out.img_b, d_b, sizeof(unsigned char)*img_out.w*img_out.h, hipMemcpyDeviceToHost);
   
    
    
    hipFree(d_r);hipFree(d_g);hipFree(d_b);hipFree(d_y);hipFree(d_cb);hipFree(d_cr);
    return img_out;
}



//Convert RGB to YUV444, all components in [0, 255]
YUV_IMG rgb2yuv(PPM_IMG img_in)
{
    YUV_IMG img_out;
    int i;//, j;
    unsigned char r, g, b;
    unsigned char y, cb, cr;
    
    img_out.w = img_in.w;
    img_out.h = img_in.h;
    img_out.img_y = (unsigned char *)malloc(sizeof(unsigned char)*img_out.w*img_out.h);
    img_out.img_u = (unsigned char *)malloc(sizeof(unsigned char)*img_out.w*img_out.h);
    img_out.img_v = (unsigned char *)malloc(sizeof(unsigned char)*img_out.w*img_out.h);
 
    for(i = 0; i < img_out.w*img_out.h; i ++){
        r = img_in.img_r[i];
        g = img_in.img_g[i];
        b = img_in.img_b[i];
        
        y  = (unsigned char)( 0.299*r + 0.587*g +  0.114*b);
        cb = (unsigned char)(-0.169*r - 0.331*g +  0.499*b + 128);
        cr = (unsigned char)( 0.499*r - 0.418*g - 0.0813*b + 128);
        
        img_out.img_y[i] = y;
        img_out.img_u[i] = cb;
        img_out.img_v[i] = cr;
    }
    
    return img_out;
}



unsigned char clip_rgb(int x)
{
    if(x > 255)
        return 255;
    if(x < 0)
        return 0;

    return (unsigned char)x;
}



//Convert YUV to RGB, all components in [0, 255]
PPM_IMG yuv2rgb(YUV_IMG img_in)
{
    PPM_IMG img_out;
    int i;
    int  rt,gt,bt;
    int y, cb, cr;
    
    
    img_out.w = img_in.w;
    img_out.h = img_in.h;
    img_out.img_r = (unsigned char *)malloc(sizeof(unsigned char)*img_out.w*img_out.h);
    img_out.img_g = (unsigned char *)malloc(sizeof(unsigned char)*img_out.w*img_out.h);
    img_out.img_b = (unsigned char *)malloc(sizeof(unsigned char)*img_out.w*img_out.h);

    for(i = 0; i < img_out.w*img_out.h; i ++){
        y  = (int)img_in.img_y[i];
        cb = (int)img_in.img_u[i] - 128;
        cr = (int)img_in.img_v[i] - 128;
        
        rt  = (int)( y + 1.402*cr);
        gt  = (int)( y - 0.344*cb - 0.714*cr); 
        bt  = (int)( y + 1.772*cb);

        img_out.img_r[i] = clip_rgb(rt);
        img_out.img_g[i] = clip_rgb(gt);
        img_out.img_b[i] = clip_rgb(bt);
    }
    
    return img_out;
}
